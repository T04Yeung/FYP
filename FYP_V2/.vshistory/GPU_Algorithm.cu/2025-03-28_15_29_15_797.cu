#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <stack>
#include <unordered_map>
#include <climits>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/partition.h>
#include <thrust/execution_policy.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "GPU_Algorithm.cuh"

__global__ void dijkstraKernel(const int* adjMatrix, int* distMatrix, int numOfVertices) {
    int src = blockIdx.x * blockDim.x + threadIdx.x;
    if (src >= numOfVertices) return;

    int* dist = new int[numOfVertices];
    bool* visited = new bool[numOfVertices];

    for (int i = 0; i < numOfVertices; i++) {
        dist[i] = INT_MAX;
        visited[i] = false;
    }

    dist[src] = 0;

    for (int i = 0; i < numOfVertices; i++) {
        int currentMinDist = INT_MAX;
        int currentVertexIdx = -1;

        for (int j = 0; j < numOfVertices; j++) {
            if (!visited[j] && dist[j] < currentMinDist) {
                currentMinDist = dist[j];
                currentVertexIdx = j;
            }
        }
        if (currentVertexIdx == -1) break;
        visited[currentVertexIdx] = true;

        for (int k = 0; k < numOfVertices; k++) {
            int weight = adjMatrix[currentVertexIdx * numOfVertices + k];
            if (weight != 0 && !visited[k] && dist[currentVertexIdx] != INT_MAX) {
				dist[k] = dist[k] < dist[currentVertexIdx] + weight ? dist[k] : dist[currentVertexIdx] + weight;
            }
        }
    }

    for (int i = 0; i < numOfVertices; i++) {
        distMatrix[src * numOfVertices + i] = dist[i];
    }

    delete[] dist;
    delete[] visited;
}

std::vector<std::vector<int>> GPU_Algorithm::GPU_Parallel_Dijkstra(const Graph& graph) {
    int numOfVertices = graph.getNumOfVertices();
    int** h_adjMatrix = graph.getAdjacencyMatrix();

    std::vector<int> _1DAdjMatrix(numOfVertices * numOfVertices);
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            _1DAdjMatrix[i * numOfVertices + j] = h_adjMatrix[i][j];
        }
    }

    int* d_AdjMatrix;
	int* d_Dist;
    hipMalloc(&d_AdjMatrix, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_Dist, numOfVertices * numOfVertices * sizeof(int));
    hipMemcpy(d_AdjMatrix, _1DAdjMatrix.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);

    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        dijkstraKernel,
        0,
        0
    );

    int gridSize = (numOfVertices + blockSize - 1) / blockSize;

    dijkstraKernel<<<gridSize, blockSize>>> (d_AdjMatrix, d_Dist, numOfVertices);

    std::vector<int> h_Dist(numOfVertices * numOfVertices);
    hipMemcpy(h_Dist.data(), d_Dist, numOfVertices * numOfVertices * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<std::vector<int>> allDistances(numOfVertices, std::vector<int>(numOfVertices));
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            allDistances[i][j] = h_Dist[i * numOfVertices + j];
        }
    }

    hipFree(d_AdjMatrix);
    hipFree(d_Dist);

    return allDistances;
}

__global__ void floydWarshallKernel(int* _1DadjMatrix, int numOfVertices, int k) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numOfVertices || j >= numOfVertices) return;

    int ij = i * numOfVertices + j;
    int ik = i * numOfVertices + k;
    int kj = k * numOfVertices + j;

    int dist_ik = _1DadjMatrix[ik];
    int dist_kj = _1DadjMatrix[kj];
    int current_dist = _1DadjMatrix[ij];

    if (dist_ik != INT_MAX && dist_kj != INT_MAX) {
        int new_dist = dist_ik + dist_kj;
        if (current_dist > new_dist) {
            _1DadjMatrix[ij] = new_dist;
        }
    }
}

std::vector<std::vector<int>> GPU_Algorithm::GPU_2DBlockMappedFW(const Graph& graph) {
	int numOfVertices = graph.getNumOfVertices();
	int** adjMatrix = graph.getAdjacencyMatrix();

	std::vector<int> _1DAdjMatrix(numOfVertices * numOfVertices);
	for (int i = 0; i < numOfVertices; i++) {
		for (int j = 0; j < numOfVertices; j++) {
			_1DAdjMatrix[i * numOfVertices + j] = (i == j) ? 0 : (adjMatrix[i][j] ? adjMatrix[i][j] : INT_MAX);
		}
	}

    int* d_adjMatrix;
    hipMalloc(&d_adjMatrix, numOfVertices * numOfVertices * sizeof(int));
    hipMemcpy(d_adjMatrix, _1DAdjMatrix.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);

	const int block_Size = 32;

	dim3 blockSize(block_Size, block_Size);
	dim3 grid = dim3((numOfVertices + block_Size - 1) / block_Size, (numOfVertices + block_Size - 1) / block_Size);

    for (int k = 0; k < numOfVertices; k++) {
		floydWarshallKernel <<<grid, blockSize>>> (d_adjMatrix, numOfVertices, k);
    }

    hipMemcpy(_1DAdjMatrix.data(), d_adjMatrix, numOfVertices * numOfVertices * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_adjMatrix);

    std::vector<std::vector<int>> result(numOfVertices, std::vector<int>(numOfVertices));
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            result[i][j] = _1DAdjMatrix[i * numOfVertices + j];
        }
    }

    return result;
}

__global__ void bfs2DKernel(
    const int* adjMatrix,
    int* depth,
    const int2* subMatrixIdx,
    int numOfVertices,
    int width,
    int blockSize,
    int* currentFrontier,
    int* currentFrontierSize,
    int* nextFrontier,
    int* nextFrontierSize
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= *currentFrontierSize) return;

    int current_vertex = currentFrontier[idx];
    int2 sub_matrix_idx = subMatrixIdx[current_vertex];

    for (int v = sub_matrix_idx.x * blockSize; v < (sub_matrix_idx.x + 1) * blockSize; v++) {
        for (int w = sub_matrix_idx.y * blockSize; w < (sub_matrix_idx.y + 1) * blockSize; w++) {
            int neighbor = v * width + w;
            if (neighbor >= numOfVertices) continue;

            if (adjMatrix[current_vertex * numOfVertices + neighbor] &&
                atomicCAS(&depth[neighbor], -1, depth[current_vertex] + 1) == -1) {
                int pos = atomicAdd(nextFrontierSize, 1);
                nextFrontier[pos] = neighbor;
            }
        }
    }
}

std::vector<int> GPU_Algorithm::GPU_2DPartition_BFS(const Graph& graph, int src) {
    int numOfVertices = graph.getNumOfVertices();
    int** h_adjMatrix = graph.getAdjacencyMatrix();
    int blockSize = 32;
    int width = (int)sqrt(numOfVertices);

    std::vector<int> _1DAdjMatrix(numOfVertices * numOfVertices);
    std::vector<int2> subMatrixIdx(numOfVertices);
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            _1DAdjMatrix[i * numOfVertices + j] = h_adjMatrix[i][j];
        }
        int row = i / width;
        int col = i % width;
        subMatrixIdx[i] = make_int2(row / blockSize, col / blockSize);
    }

    int* d_adj, * d_depth, * d_currentFrontier, * d_nextFrontier;
    int2* d_subMatrixIdx;
    int* d_currentSize, * d_nextSize;

    hipMalloc(&d_adj, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_depth, numOfVertices * sizeof(int));
    hipMalloc(&d_subMatrixIdx, numOfVertices * sizeof(int2));
    hipMalloc(&d_currentFrontier, numOfVertices * sizeof(int));
    hipMalloc(&d_nextFrontier, numOfVertices * sizeof(int));
    hipMalloc(&d_currentSize, sizeof(int));
    hipMalloc(&d_nextSize, sizeof(int));

    std::vector<int> h_depth(numOfVertices, -1);
    h_depth[src] = 0;
    std::vector<int> h_frontier = {src};

    hipMemcpy(d_adj, _1DAdjMatrix.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_depth, h_depth.data(), numOfVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_subMatrixIdx, subMatrixIdx.data(), numOfVertices * sizeof(int2), hipMemcpyHostToDevice);

    int currentSize = 1;
    int nextSize = 0;

    do {
        hipMemcpy(d_currentFrontier, h_frontier.data(), currentSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_currentSize, &currentSize, sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_nextSize, 0, sizeof(int));

        int gridSize = (currentSize + blockSize - 1) / blockSize;
        bfs2DKernel <<<gridSize, blockSize>>> (d_adj, d_depth, d_subMatrixIdx, numOfVertices, width,
            blockSize, d_currentFrontier, d_currentSize,
            d_nextFrontier, d_nextSize);

        hipMemcpy(&nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost);

        std::swap(d_currentFrontier, d_nextFrontier);
        currentSize = nextSize;
        nextSize = 0;

        h_frontier.resize(currentSize);
        hipMemcpy(h_frontier.data(), d_currentFrontier, currentSize * sizeof(int), hipMemcpyDeviceToHost);

    } while (currentSize > 0);

    hipMemcpy(h_depth.data(), d_depth, numOfVertices * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_adj);
    hipFree(d_depth);
    hipFree(d_subMatrixIdx);
    hipFree(d_currentFrontier);
    hipFree(d_nextFrontier);
    hipFree(d_currentSize);
    hipFree(d_nextSize);

    return h_depth;
}

__device__ int find(int element, int* parent) {
    while (parent[element] != element) {
        int p = parent[element];
        parent[element] = parent[p];
        element = parent[element];
    }
    return element;
}

__device__ bool unite(int setA, int setB, int* parent, int* rank) {
    int rootA = find(setA, parent);
    int rootB = find(setB, parent);
    if (rootA == rootB) return false;

    if (rank[rootA] < rank[rootB]) {
        atomicExch(&parent[rootA], rootB);
    }
    else {
        atomicExch(&parent[rootB], rootA);
        if (rank[rootA] == rank[rootB])
            atomicAdd(&rank[rootA], 1);
    }
    return true;
}

__global__ void filterKruskalKernel(Edge* edges, int numOfEdges, int* parent, int* rank, Edge* mst, int* mstSize, int medianWeight) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numOfEdges) return;

    Edge edge = edges[idx];
    if (edge.weight > medianWeight) return;

    int rootA = find(edge.from, parent);
    int rootB = find(edge.to, parent);

    if (rootA != rootB && unite(rootA, rootB, parent, rank)) {
        int pos = atomicAdd(mstSize, 1);
        mst[pos] = edge;
    }
}

std::vector<GPU_Edge> GPU_Algorithm::GPU_Parallel_FilterKruskal(Graph& graph) {
	std::vector<Edge> edges = graph.getUndirectedEdges();
	std::vector<GPU_Edge> h_edges(edges.size());
    for (auto& edge : edges) {
		h_edges.push_back(GPU_Edge(edge.id, edge.from, edge.to, edge.weight));
    }

	int numOfVertices = graph.getNumOfVertices();
	int numOfEdges = h_edges.size();

    GPU_Edge* d_edges;
    int* d_parent, * d_rank, * d_mstSize;
    GPU_Edge* d_mstEdges;

    hipMalloc(&d_edges, numOfEdges * sizeof(GPU_Edge));
    hipMalloc(&d_parent, numOfVertices * sizeof(int));
    hipMalloc(&d_rank, numOfVertices * sizeof(int));
    hipMalloc(&d_mstEdges, numOfEdges * sizeof(GPU_Edge));
    hipMalloc(&d_mstSize, sizeof(int));

    hipMemcpy(d_edges, h_edges.data(), numOfEdges * sizeof(GPU_Edge), hipMemcpyHostToDevice);
    hipMemset(d_rank, 0, numOfVertices * sizeof(int));
    hipMemset(d_mstSize, 0, sizeof(int));

	thrust::sequence(thrust::device, d_parent, d_parent + numOfVertices);

    thrust::sort(thrust::device, d_edges, d_edges + numOfEdges, [=] __host__ __device__(const GPU_Edge & e1, const GPU_Edge & e2) {
		return e1.weight < e2.weight;
    });

	int remainingEdges = numOfEdges;

    while (remainingEdges > 0) {
        GPU_Edge medianEdge;
        hipMemcpy(&medianEdge, d_edges + remainingEdges / 2, sizeof(GPU_Edge), hipMemcpyDeviceToHost);

		int medianWeight = medianEdge.weight;

        int blockSize = 32;
		int gridSize = (remainingEdges + blockSize - 1) / blockSize;

        filterKruskalKernel<<<gridSize, blockSize>>> (d_edges, remainingEdges, d_parent, d_rank, d_mstEdges, d_mstSize, medianWeight);
        hipDeviceSynchronize();

		auto endOfEdges = thrust::partition(thrust::device, d_edges, d_edges + remainingEdges, [medianWeight] __host__ __device__(const GPU_Edge & e) {
			return e.weight <= medianWeight;
		});
		remainingEdges = (d_edges + remainingEdges) - endOfEdges;
    }

    int finalSize;
    hipMemcpy(&finalSize, d_mstSize, sizeof(int), hipMemcpyDeviceToHost);
    std::vector<GPU_Edge> mst(finalSize);
    hipMemcpy(mst.data(), d_mstEdges, finalSize * sizeof(GPU_Edge), hipMemcpyDeviceToHost);

    hipFree(d_edges);
    hipFree(d_parent);
    hipFree(d_rank);
    hipFree(d_mstEdges);
    hipFree(d_mstSize);

    return mst;
}