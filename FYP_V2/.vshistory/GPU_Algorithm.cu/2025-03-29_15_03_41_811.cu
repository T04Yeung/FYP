#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <stack>
#include <unordered_map>
#include <climits>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/partition.h>
#include <thrust/execution_policy.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "GPU_Algorithm.cuh"

__global__ void dijkstraKernel(const int* adjMatrix, int* distMatrix, int numOfVertices) {
    int src = blockIdx.x * blockDim.x + threadIdx.x;
    if (src >= numOfVertices) return;

    int* dist = new int[numOfVertices];
    bool* visited = new bool[numOfVertices];

    for (int i = 0; i < numOfVertices; i++) {
        dist[i] = INT_MAX;
        visited[i] = false;
    }

    dist[src] = 0;

    for (int i = 0; i < numOfVertices; i++) {
        int currentMinDist = INT_MAX;
        int currentVertexIdx = -1;

        for (int j = 0; j < numOfVertices; j++) {
            if (!visited[j] && dist[j] < currentMinDist) {
                currentMinDist = dist[j];
                currentVertexIdx = j;
            }
        }
        if (currentVertexIdx == -1) break;
        visited[currentVertexIdx] = true;

        for (int k = 0; k < numOfVertices; k++) {
            int weight = adjMatrix[currentVertexIdx * numOfVertices + k];
            if (weight != 0 && !visited[k] && dist[currentVertexIdx] != INT_MAX) {
				dist[k] = dist[k] < dist[currentVertexIdx] + weight ? dist[k] : dist[currentVertexIdx] + weight;
            }
        }
    }

    for (int i = 0; i < numOfVertices; i++) {
        distMatrix[src * numOfVertices + i] = dist[i];
    }

    delete[] dist;
    delete[] visited;
}

std::vector<std::vector<int>> GPU_Algorithm::GPU_Parallel_Dijkstra(const Graph& graph) {
    int numOfVertices = graph.getNumOfVertices();
    int** h_adjMatrix = graph.getAdjacencyMatrix();

    std::vector<int> _1DAdjMatrix(numOfVertices * numOfVertices);
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            _1DAdjMatrix[i * numOfVertices + j] = h_adjMatrix[i][j];
        }
    }

    int* d_AdjMatrix;
	int* d_Dist;
    hipMalloc(&d_AdjMatrix, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_Dist, numOfVertices * numOfVertices * sizeof(int));
    hipMemcpy(d_AdjMatrix, _1DAdjMatrix.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);

    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        dijkstraKernel,
        0,
        0
    );

    int gridSize = (numOfVertices + blockSize - 1) / blockSize;

    dijkstraKernel<<<gridSize, blockSize>>> (d_AdjMatrix, d_Dist, numOfVertices);

    std::vector<int> h_Dist(numOfVertices * numOfVertices);
    hipMemcpy(h_Dist.data(), d_Dist, numOfVertices * numOfVertices * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<std::vector<int>> allDistances(numOfVertices, std::vector<int>(numOfVertices));
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            allDistances[i][j] = h_Dist[i * numOfVertices + j];
        }
    }

    hipFree(d_AdjMatrix);
    hipFree(d_Dist);

    return allDistances;
}

__global__ void floydWarshallKernel(int* _1DadjMatrix, int numOfVertices, int k) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numOfVertices || j >= numOfVertices) return;

    int ij = i * numOfVertices + j;
    int ik = i * numOfVertices + k;
    int kj = k * numOfVertices + j;

    int dist_ik = _1DadjMatrix[ik];
    int dist_kj = _1DadjMatrix[kj];
    int current_dist = _1DadjMatrix[ij];

    if (dist_ik != INT_MAX && dist_kj != INT_MAX) {
        int new_dist = dist_ik + dist_kj;
        if (current_dist > new_dist) {
            _1DadjMatrix[ij] = new_dist;
        }
    }
}

std::vector<std::vector<int>> GPU_Algorithm::GPU_2DBlockMappedFW(const Graph& graph) {
	int numOfVertices = graph.getNumOfVertices();
	int** adjMatrix = graph.getAdjacencyMatrix();

	std::vector<int> _1DAdjMatrix(numOfVertices * numOfVertices);
	for (int i = 0; i < numOfVertices; i++) {
		for (int j = 0; j < numOfVertices; j++) {
			_1DAdjMatrix[i * numOfVertices + j] = (i == j) ? 0 : (adjMatrix[i][j] ? adjMatrix[i][j] : INT_MAX);
		}
	}

    int* d_adjMatrix;
    hipMalloc(&d_adjMatrix, numOfVertices * numOfVertices * sizeof(int));
    hipMemcpy(d_adjMatrix, _1DAdjMatrix.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);

	const int block_Size = 32;

	dim3 blockSize(block_Size, block_Size);
	dim3 grid = dim3((numOfVertices + block_Size - 1) / block_Size, (numOfVertices + block_Size - 1) / block_Size);

    for (int k = 0; k < numOfVertices; k++) {
		floydWarshallKernel <<<grid, blockSize>>> (d_adjMatrix, numOfVertices, k);
    }

    hipMemcpy(_1DAdjMatrix.data(), d_adjMatrix, numOfVertices * numOfVertices * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_adjMatrix);

    std::vector<std::vector<int>> result(numOfVertices, std::vector<int>(numOfVertices));
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            result[i][j] = _1DAdjMatrix[i * numOfVertices + j];
        }
    }

    return result;
}

__global__ void bfs2DKernel(
    const int* adjMatrix,
    int* depth,
    const int2* subMatrixIdx,
    int numOfVertices,
    int width,
    int blockSize,
    int* currentFrontier,
    int* currentFrontierSize,
    int* nextFrontier,
    int* nextFrontierSize
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= *currentFrontierSize) return;

    int current_vertex = currentFrontier[idx];
    int2 sub_matrix_idx = subMatrixIdx[current_vertex];

    for (int v = sub_matrix_idx.x * blockSize; v < (sub_matrix_idx.x + 1) * blockSize; v++) {
        for (int w = sub_matrix_idx.y * blockSize; w < (sub_matrix_idx.y + 1) * blockSize; w++) {
            int neighbor = v * width + w;
            if (neighbor >= numOfVertices) continue;

            if (adjMatrix[current_vertex * numOfVertices + neighbor] &&
                atomicCAS(&depth[neighbor], -1, depth[current_vertex] + 1) == -1) {
                int pos = atomicAdd(nextFrontierSize, 1);
                nextFrontier[pos] = neighbor;
            }
        }
    }
}

std::vector<int> GPU_Algorithm::GPU_2DPartition_BFS(const Graph& graph, int src) {
    int numOfVertices = graph.getNumOfVertices();
    int** h_adjMatrix = graph.getAdjacencyMatrix();
    int blockSize = 32;
    int width = (int)sqrt(numOfVertices);

    std::vector<int> _1DAdjMatrix(numOfVertices * numOfVertices);
    std::vector<int2> subMatrixIdx(numOfVertices);
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            _1DAdjMatrix[i * numOfVertices + j] = h_adjMatrix[i][j];
        }
        int row = i / width;
        int col = i % width;
        subMatrixIdx[i] = make_int2(row / blockSize, col / blockSize);
    }

    int* d_adj, * d_depth, * d_currentFrontier, * d_nextFrontier;
    int2* d_subMatrixIdx;
    int* d_currentSize, * d_nextSize;

    hipMalloc(&d_adj, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_depth, numOfVertices * sizeof(int));
    hipMalloc(&d_subMatrixIdx, numOfVertices * sizeof(int2));
    hipMalloc(&d_currentFrontier, numOfVertices * sizeof(int));
    hipMalloc(&d_nextFrontier, numOfVertices * sizeof(int));
    hipMalloc(&d_currentSize, sizeof(int));
    hipMalloc(&d_nextSize, sizeof(int));

    std::vector<int> h_depth(numOfVertices, -1);
    h_depth[src] = 0;
    std::vector<int> h_frontier = {src};

    hipMemcpy(d_adj, _1DAdjMatrix.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_depth, h_depth.data(), numOfVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_subMatrixIdx, subMatrixIdx.data(), numOfVertices * sizeof(int2), hipMemcpyHostToDevice);

    int currentSize = 1;
    int nextSize = 0;

    do {
        hipMemcpy(d_currentFrontier, h_frontier.data(), currentSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_currentSize, &currentSize, sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_nextSize, 0, sizeof(int));

        int gridSize = (currentSize + blockSize - 1) / blockSize;
        bfs2DKernel <<<gridSize, blockSize>>> (d_adj, d_depth, d_subMatrixIdx, numOfVertices, width,
            blockSize, d_currentFrontier, d_currentSize,
            d_nextFrontier, d_nextSize);

        hipMemcpy(&nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost);

        std::swap(d_currentFrontier, d_nextFrontier);
        currentSize = nextSize;
        nextSize = 0;

        h_frontier.resize(currentSize);
        hipMemcpy(h_frontier.data(), d_currentFrontier, currentSize * sizeof(int), hipMemcpyDeviceToHost);

    } while (currentSize > 0);

    hipMemcpy(h_depth.data(), d_depth, numOfVertices * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_adj);
    hipFree(d_depth);
    hipFree(d_subMatrixIdx);
    hipFree(d_currentFrontier);
    hipFree(d_nextFrontier);
    hipFree(d_currentSize);
    hipFree(d_nextSize);

    return h_depth;
}

__host__ __device__ int find(int element, int* parent) {
    while (parent[element] != element) {
        int p = parent[element];
        parent[element] = parent[p];
        element = parent[element];
    }
    return element;
}

__device__ bool unite(int setA, int setB, int* parent, int* rank) {
    int rootA = find(setA, parent);
    int rootB = find(setB, parent);
    if (rootA == rootB) return false;

    if (rank[rootA] < rank[rootB]) {
        atomicExch(&parent[rootA], rootB);
    }
    else {
        atomicExch(&parent[rootB], rootA);
        if (rank[rootA] == rank[rootB])
            atomicAdd(&rank[rootA], 1);
    }
    return true;
}

__global__ void filterKruskalKernel(Edge* edges, int numOfEdges, int* parent, int* rank, Edge* mst, int* mstSize, int medianWeight, int* edgeAdded) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numOfEdges) return;

    Edge edge = edges[idx];
    if (edge.weight > medianWeight) return;

    int rootA = find(edge.from, parent);
    int rootB = find(edge.to, parent);

    if (rootA != rootB && unite(rootA, rootB, parent, rank)) {
        if (!atomicExch(&edgeAdded[idx], 1)) {
            int pos = atomicAdd(mstSize, 1);
            mst[pos] = edge;
        }
    }
}

std::vector<Edge> GPU_Algorithm::GPU_Parallel_FilterKruskal(Graph& graph) {
    std::vector<Edge> edges = graph.getUndirectedEdges();

    int numOfVertices = graph.getNumOfVertices();
    int numOfEdges = edges.size();

    Edge* d_edges;
    int* d_parent, * d_rank, * d_mstSize;
    Edge* d_mstEdges;
    int* d_edgeAdded;

    hipMalloc(&d_edges, numOfEdges * sizeof(Edge));
    hipMalloc(&d_parent, numOfVertices * sizeof(int));
    hipMalloc(&d_rank, numOfVertices * sizeof(int));
    hipMalloc(&d_mstEdges, numOfEdges * sizeof(Edge));
    hipMalloc(&d_mstSize, sizeof(int));
    hipMalloc(&d_edgeAdded, numOfEdges * sizeof(int));

    hipMemcpy(d_edges, edges.data(), numOfEdges * sizeof(Edge), hipMemcpyHostToDevice);
    hipMemset(d_rank, 0, numOfVertices * sizeof(int));
    hipMemset(d_mstSize, 0, sizeof(int));
    hipMemset(d_edgeAdded, 0, numOfEdges * sizeof(int));

    thrust::sequence(thrust::device, d_parent, d_parent + numOfVertices);

    thrust::sort(thrust::device, d_edges, d_edges + numOfEdges, [=] __host__ __device__(const Edge & e1, const Edge & e2) {
        return e1.weight < e2.weight;
    });

    int remainingEdges = numOfEdges;

    while (remainingEdges > 0) {
        Edge medianEdge;
        hipMemcpy(&medianEdge, d_edges + remainingEdges / 2, sizeof(Edge), hipMemcpyDeviceToHost);

        int medianWeight = medianEdge.weight;

        int blockSize = 32;
        int gridSize = (remainingEdges + blockSize - 1) / blockSize;

        filterKruskalKernel <<<gridSize, blockSize >>> (d_edges, remainingEdges, d_parent, d_rank, d_mstEdges, d_mstSize, medianWeight, d_edgeAdded);
        hipDeviceSynchronize();

        auto newEnd = thrust::partition(thrust::device, d_edges, d_edges + remainingEdges,
            [d_parent] __host__ __device__(const Edge & e) {
            int rootA = find(e.from, d_parent);
            int rootB = find(e.to, d_parent);
            return rootA != rootB;
        });
        remainingEdges = newEnd - d_edges;
    }

    int finalSize;
    hipMemcpy(&finalSize, d_mstSize, sizeof(int), hipMemcpyDeviceToHost);
    std::vector<Edge> mst(finalSize);
    hipMemcpy(mst.data(), d_mstEdges, finalSize * sizeof(Edge), hipMemcpyDeviceToHost);

    hipFree(d_edges);
    hipFree(d_parent);
    hipFree(d_rank);
    hipFree(d_mstEdges);
    hipFree(d_mstSize);
    hipFree(d_edgeAdded);

    return mst;
}

__global__ void matrixMultiplyKernel(const int* A, const int* B, int* C, int numOfVertices) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= numOfVertices || col >= numOfVertices) return;

    int sum = 0;
    for (int k = 0; k < numOfVertices; ++k)
        sum += A[row * numOfVertices + k] * B[k * numOfVertices + col];
    C[row * numOfVertices + col] = sum;
}

__global__ void traceKernel(const int* mat, int* count, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) atomicAdd(count, mat[idx * n + idx]);
}

int GPU_Algorithm::GPU_Triangle_Counting(Graph& graph) {
	int numOfVertices = graph.getNumOfVertices();
	int** adjMatrix = graph.getAdjacencyMatrix();

    std::vector<int> h_A(numOfVertices * numOfVertices);
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            h_A[i * numOfVertices + j] = (adjMatrix[i][j] != 0 || adjMatrix[j][i] != 0) ? 1 : 0;
        }
    }

	int* d_A, * d_A2, * d_A3, *d_count;
    hipMalloc(&d_A, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_A2, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_A3, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_count, sizeof(int));

    hipMemcpy(d_A, h_A.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);
    dim3 blockSize(32, 32);
    dim3 grid((numOfVertices + blockSize.x - 1) / blockSize.x, (numOfVertices + blockSize.y - 1) / blockSize.y);
    matrixMultiplyKernel <<<grid, blockSize>>> (d_A, d_A, d_A2, numOfVertices);
    matrixMultiplyKernel <<<grid, blockSize>>> (d_A2, d_A, d_A3, numOfVertices);

    int h_count = 0;
    hipMemcpy(d_count, &h_count, sizeof(int), hipMemcpyHostToDevice);
	int blockSize2 = 256;
	int grid2 = (numOfVertices + blockSize2 - 1) / blockSize2;
    traceKernel <<<grid2, blockSize2>>> (d_A3, d_count, numOfVertices);
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_A2);
    hipFree(d_A3);
    hipFree(d_count);

	return h_count / 6;
}