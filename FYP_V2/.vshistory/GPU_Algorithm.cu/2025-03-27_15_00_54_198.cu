#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <stack>
#include <unordered_map>
#include <climits>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "GPU_Algorithm.cuh"

__global__ void dijkstraKernel(const int* adjMatrix, int* distMatrix, int numOfVertices) {
    int src = blockIdx.x * blockDim.x + threadIdx.x;
    if (src >= numOfVertices) return;

    int* dist = new int[numOfVertices];
    bool* visited = new bool[numOfVertices];

    for (int i = 0; i < numOfVertices; i++) {
        dist[i] = INT_MAX;
        visited[i] = false;
    }

    dist[src] = 0;

    for (int i = 0; i < numOfVertices; i++) {
        int currentMinDist = INT_MAX;
        int currentVertexIdx = -1;

        for (int j = 0; j < numOfVertices; j++) {
            if (!visited[j] && dist[j] < currentMinDist) {
                currentMinDist = dist[j];
                currentVertexIdx = j;
            }
        }
        if (currentVertexIdx == -1) break;
        visited[currentVertexIdx] = true;

        for (int k = 0; k < numOfVertices; k++) {
            int weight = adjMatrix[currentVertexIdx * numOfVertices + k];
            if (weight != 0 && !visited[k] && dist[currentVertexIdx] != INT_MAX) {
				dist[k] = dist[k] < dist[currentVertexIdx] + weight ? dist[k] : dist[currentVertexIdx] + weight;
            }
        }
    }

    for (int i = 0; i < numOfVertices; i++) {
        distMatrix[src * numOfVertices + i] = dist[i];
    }

    delete[] dist;
    delete[] visited;
}

std::vector<std::vector<int>> GPU_Algorithm::GPU_Parallel_Dijkstra(const Graph& graph) {
    int numOfVertices = graph.getNumOfVertices();
    int** h_adjMatrix = graph.getAdjacencyMatrix();

    std::vector<int> _1DAdjMatrix(numOfVertices * numOfVertices);
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            _1DAdjMatrix[i * numOfVertices + j] = h_adjMatrix[i][j];
        }
    }

    int* d_AdjMatrix;
	int* d_Dist;
    hipMalloc(&d_AdjMatrix, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_Dist, numOfVertices * numOfVertices * sizeof(int));
    hipMemcpy(d_AdjMatrix, _1DAdjMatrix.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);

    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        dijkstraKernel,
        0,
        0
    );

    int gridSize = (numOfVertices + blockSize - 1) / blockSize;

    dijkstraKernel<<<gridSize, blockSize>>> (d_AdjMatrix, d_Dist, numOfVertices);

    std::vector<int> h_Dist(numOfVertices * numOfVertices);
    hipMemcpy(h_Dist.data(), d_Dist, numOfVertices * numOfVertices * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<std::vector<int>> allDistances(numOfVertices, std::vector<int>(numOfVertices));
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            allDistances[i][j] = h_Dist[i * numOfVertices + j];
        }
    }

    hipFree(d_AdjMatrix);
    hipFree(d_Dist);

    return allDistances;
}

__global__ void bfs2DKernel(
    const int* adjMatrix,
    int* depth,
    const int2* subMatrixIdx,
    int numOfVertices,
    int width,
    int blockSize,
    int* currentFrontier,
    int* currentFrontierSize,
    int* nextFrontier,
    int* nextFrontierSize
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= *currentFrontierSize) return;

    int current_vertex = currentFrontier[idx];
    int2 sub_matrix_idx = subMatrixIdx[current_vertex];

    for (int v = sub_matrix_idx.x * blockSize; v < (sub_matrix_idx.x + 1) * blockSize; v++) {
        for (int w = sub_matrix_idx.y * blockSize; w < (sub_matrix_idx.y + 1) * blockSize; w++) {
            int neighbor = v * width + w;
            if (neighbor >= numOfVertices) continue;

            if (adjMatrix[current_vertex * numOfVertices + neighbor] &&
                atomicCAS(&depth[neighbor], -1, depth[current_vertex] + 1) == -1) {
                int pos = atomicAdd(nextFrontierSize, 1);
                nextFrontier[pos] = neighbor;
            }
        }
    }
}

std::vector<int> GPU_Algorithm::GPU_2DPartition_BFS(const Graph& graph, int src) {
    int numOfVertices = graph.getNumOfVertices();
    int** h_adjMatrix = graph.getAdjacencyMatrix();
    int blockSize = 32;
    int width = (int)sqrt(numOfVertices);

    std::vector<int> _1DAdjMatrix(numOfVertices * numOfVertices);
    std::vector<int2> subMatrixIdx(numOfVertices);
    for (int i = 0; i < numOfVertices; i++) {
        for (int j = 0; j < numOfVertices; j++) {
            _1DAdjMatrix[i * numOfVertices + j] = h_adjMatrix[i][j];
        }
        int row = i / width;
        int col = i % width;
        subMatrixIdx[i] = make_int2(row / blockSize, col / blockSize);
    }

    int* d_adj, * d_depth, * d_currentFrontier, * d_nextFrontier;
    int2* d_subMatrixIdx;
    int* d_currentSize, * d_nextSize;

    hipMalloc(&d_adj, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_depth, numOfVertices * sizeof(int));
    hipMalloc(&d_subMatrixIdx, numOfVertices * sizeof(int2));
    hipMalloc(&d_currentFrontier, numOfVertices * sizeof(int));
    hipMalloc(&d_nextFrontier, numOfVertices * sizeof(int));
    hipMalloc(&d_currentSize, sizeof(int));
    hipMalloc(&d_nextSize, sizeof(int));

    std::vector<int> h_depth(numOfVertices, -1);
    h_depth[src] = 0;
    std::vector<int> h_frontier = {src};

    hipMemcpy(d_adj, _1DAdjMatrix.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_depth, h_depth.data(), numOfVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_subMatrixIdx, subMatrixIdx.data(), numOfVertices * sizeof(int2), hipMemcpyHostToDevice);

    int currentSize = 1;
    int nextSize = 0;

    do {
        hipMemcpy(d_currentFrontier, h_frontier.data(), currentSize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_currentSize, &currentSize, sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_nextSize, 0, sizeof(int));

        int blockSize, minGridSize;
        hipOccupancyMaxPotentialBlockSize(
            &minGridSize,
            &blockSize,
            bfs2DKernel,
            0,
            0
        );

        int gridSize = (currentSize + blockSize - 1) / blockSize;
        bfs2DKernel <<<gridSize, blockSize>>> (d_adj, d_depth, d_subMatrixIdx, numOfVertices, width,
            blockSize, d_currentFrontier, d_currentSize,
            d_nextFrontier, d_nextSize);

        hipMemcpy(&nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost);

        std::swap(d_currentFrontier, d_nextFrontier);
        currentSize = nextSize;
        nextSize = 0;

        h_frontier.resize(currentSize);
        hipMemcpy(h_frontier.data(), d_currentFrontier, currentSize * sizeof(int), hipMemcpyDeviceToHost);

    } while (currentSize > 0);

    hipMemcpy(h_depth.data(), d_depth, numOfVertices * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_adj);
    hipFree(d_depth);
    hipFree(d_subMatrixIdx);
    hipFree(d_currentFrontier);
    hipFree(d_nextFrontier);
    hipFree(d_currentSize);
    hipFree(d_nextSize);

    return h_depth;
}

std::vector<int> getCompileOrderForKosaraju(const Graph& graph, const int& numOfVertices) {
    std::vector<int> compileOrder;
    std::vector<bool> visited(numOfVertices, false);

    for (int i = 0; i < numOfVertices; i++) {
        if (!visited[i]) {
            std::stack<int> stack;
            stack.push(i);
            visited[i] = true;

            while (!stack.empty()) {
                int currentVertex = stack.top();
                bool hasUnvisitedVertex = false;
                for (int j = 0; j < numOfVertices; j++) {
                    if (graph.getAdjacencyMatrix()[currentVertex][j] != 0 && !visited[j]) {
                        stack.push(j);
                        visited[j] = true;
                        hasUnvisitedVertex = true;
                        break;
                    }
                }

                if (!hasUnvisitedVertex) {
                    stack.pop();
                    compileOrder.push_back(currentVertex);
                }
            }
        }
    }

    return compileOrder;
}

__global__ void kosarajuKernel(
    const int* adjMatrix,
    const int* compileOrder,
    bool* visited,
    int* components,
    int numVertices
) {
    int idx = numVertices - 1 - (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < 0) return;

    int currentVertex = compileOrder[idx];

    bool expected = false;
    /*if (atomicCAS(&visited[currentVertex], expected, true)) {
        int componentId = currentVertex;
        components[currentVertex] = componentId;

        int* queue = new int[numVertices];
        int front = 0, rear = 0;
        queue[rear++] = currentVertex;

        while (front < rear) {
            int currentV = queue[front++];
            for (int i = 0; i < numVertices; i++) {
                if (adjMatrix[currentV * numVertices + i] && !visited[i]) {
                    if (atomicCAS(&visited[i], false, true)) {
                        components[i] = componentId;
                        queue[rear++] = i;
                    }
                }
            }
        }
        delete[] queue;
    }*/
}

std::vector<std::vector<int>> GPU_Algorithm::GPU_Kosaraju(Graph& graph) {
	int numOfVertices = graph.getNumOfVertices();
	std::vector<int> compileOrder = getCompileOrderForKosaraju(graph, numOfVertices);

	Graph reverseGraph = graph.getReversedGraph();
    int** reversedAdjMatrix = reverseGraph.getAdjacencyMatrix();

    std::vector<int> reversed1DAdj(numOfVertices * numOfVertices);
    for (int i = 0; i < numOfVertices; ++i) {
        for (int j = 0; j < numOfVertices; ++j) {
            reversed1DAdj[i * numOfVertices + j] = reversedAdjMatrix[i][j];
        }
    }

    int* d_adj, * d_compileOrder, * d_components;
    bool* d_visited;
    hipMalloc(&d_adj, numOfVertices * numOfVertices * sizeof(int));
    hipMalloc(&d_compileOrder, numOfVertices * sizeof(int));
    hipMalloc(&d_components, numOfVertices * sizeof(int));
    hipMalloc(&d_visited, numOfVertices * sizeof(bool));

    hipMemcpy(d_adj, reversed1DAdj.data(), numOfVertices * numOfVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_compileOrder, compileOrder.data(), numOfVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_components, -1, numOfVertices * sizeof(int));
    hipMemset(d_visited, false, numOfVertices * sizeof(bool));

    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize,
        &blockSize,
        dijkstraKernel,
        0,
        0
    );

    int gridSize = (numOfVertices + blockSize - 1) / blockSize;

    kosarajuKernel <<<gridSize, blockSize >>> (d_adj, d_compileOrder, d_visited, d_components, numOfVertices);
    hipDeviceSynchronize();

    std::vector<int> h_components(numOfVertices);
    hipMemcpy(h_components.data(), d_components, numOfVertices * sizeof(int), hipMemcpyDeviceToHost);

    std::unordered_map<int, std::vector<int>> componentMap;
    for (int i = 0; i < numOfVertices; ++i) {
        componentMap[h_components[i]].push_back(i);
    }

    hipFree(d_adj);
    hipFree(d_compileOrder);
    hipFree(d_components);
    hipFree(d_visited);

    std::vector<std::vector<int>> sccs;
    for (const auto& pair : componentMap) {
        sccs.push_back(pair.second);
    }

    return sccs;
}